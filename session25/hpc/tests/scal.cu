#include "hip/hip_runtime.h"
#include <cstdlib>
#include <iostream>
#include <hpc/cuda/check.h>
#include <hpc/cuda/properties.h>

#define N 257

__global__ void axpy(std::size_t n, double alpha, double* x, double* y)
{
    std::size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        y[tid] += alpha * x[tid];
    }
}

template <typename T, typename ALPHA>
__global__ void scal(std::size_t n, ALPHA alpha, T* x)
{
    std::size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    if (tid < n) {
        x[tid] *= alpha;
    }
}

std::size_t ceildiv(std::size_t x, std::size_t y) {
   /* note that we expect x > 0 && y > 0;
      not safe against overflows but we expect y to be small */
   return (x + y - 1) / y;
}

int main() {
    double a[N];
    for (std::size_t i = 0; i < N; ++i) {
        a[i] = i;
    }

    /* transfer vectors to GPU memory */
    double* cuda_a;
    CHECK_CUDA(hipMalloc, (void**)&cuda_a, N * sizeof(double));
    CHECK_CUDA(hipMemcpy, cuda_a, a, N * sizeof(double),
               hipMemcpyHostToDevice);

    /* execute kernel function on GPU */
    std::size_t warp_size = hpc::cuda::get_warp_size(); /* typically 32 */
    std::size_t nof_warps = ceildiv(N, warp_size);
    std::size_t warps_per_block =
        hpc::cuda::get_max_threads_per_block() / warp_size / 4; /* typically 8 */
    std::size_t nof_blocks = ceildiv(nof_warps, warps_per_block);
    std::size_t threads_per_block;

    if (nof_blocks == 1) {
        threads_per_block = N;
    } else {
        threads_per_block = warps_per_block * warp_size;
    }
    //axpy<<<nof_blocks, threads_per_block>>>(N, 2.0, cuda_a, cuda_b);
    scal<<<nof_blocks, threads_per_block>>>(N,2, cuda_a);

    /* transfer result vector from GPU to host memory */
    CHECK_CUDA(hipMemcpy, a, cuda_a, N * sizeof(double),
               hipMemcpyDeviceToHost);
    /* free space allocated at GPU memory */
    CHECK_CUDA(hipFree, cuda_a);

    /* print result */
    for (std::size_t i = 0; i < N; ++i) {
        std::cout << " " << a[i];
        if (i % 10 == 0) std::cout << std::endl;
    }
    std::cout << std::endl;
}
